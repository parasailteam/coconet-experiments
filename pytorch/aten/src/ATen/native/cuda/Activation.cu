#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES

#include <ATen/native/Activation.h>

#include <cmath>

#include <thrust/tuple.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/core/Array.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/DistributionTemplates.h>
#include <ATen/native/Resize.h>
#include <c10/cuda/CUDAMathCompat.h>

namespace at {
namespace native {

// -----------------------------------
// glu forward
// -----------------------------------
void glu_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, iter.dtype(), "glu_cuda", [&]() {
    using acc_t = at::acc_type<scalar_t, true>;
    gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a_, scalar_t b_) -> scalar_t {
      const acc_t a = a_;
      const acc_t b = b_;
      const acc_t one = acc_t(1);
      const acc_t sigmoid = one / (one + std::exp(-b));
      return a * sigmoid;
    });
  });
}

// -----------------------------------
// glu backward
// -----------------------------------

// Byte offsets don't require multiplication by sizeof(T), so are slightly cheaper.
// For fixed offsets, this removes all penalty from 64-bit indexing.
template <typename T>
__device__ T* byte_offset(T* ptr, int64_t offset) {
  using byte_ptr_t = typename std::conditional<
    std::is_const<T>::value, const char*, char*>::type;
  return reinterpret_cast<T*>(
    reinterpret_cast<byte_ptr_t>(ptr) + offset
  );
}

template <typename scalar_t, typename OffsetCalc>
__global__ void glu_backward_kernel(
    int numel, scalar_t* gI, const scalar_t* I, const scalar_t* gO,
    OffsetCalc offset_calculator,
    int64_t gI_byte_offset, int64_t I_byte_offset) {
  using acc_t = at::acc_type<scalar_t, true>;

  const uint32_t linear_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (linear_index >= numel) {
    return;
  }
  const auto offsets = offset_calculator.get(linear_index);

  // We explicitly iterate over the first half of the input tensor, and
  // gI_byte_offset and I_byte_offset are the offsets to access the
  // corresponding index in the second half of the tensor.
  const acc_t a = I[offsets[1]];
  const acc_t b = *byte_offset(I + offsets[1], I_byte_offset);
  const acc_t gO_val = gO[offsets[2]];

  const auto one = acc_t(1);
  const acc_t sigmoid = one / (one + std::exp(-b));

  auto* gA = gI + offsets[0];
  *gA = sigmoid * gO_val;

  auto* gB = byte_offset(gA, gI_byte_offset);
  *gB = (one - sigmoid) * sigmoid * gO_val * a;
}

void launch_glu_backward_kernel(const TensorIteratorBase& iter,
                                int64_t gI_stride, int64_t I_stride) {
  const auto N = iter.numel();
  TORCH_INTERNAL_ASSERT_DEBUG_ONLY(N > 0 && N <= std::numeric_limits<int32_t>::max());
  const auto offset_calculator = make_element_offset_calculator<3>(iter);
  constexpr int64_t block_size = 256;
  const int64_t grid = (N + block_size - 1) / block_size;
  const auto stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "glu_backward_cuda", [&] {
    auto gI = static_cast<scalar_t*>(iter.data_ptr(0));
    auto I = static_cast<const scalar_t*>(iter.data_ptr(1));
    auto gO = static_cast<const scalar_t*>(iter.data_ptr(2));
    glu_backward_kernel<<<grid, block_size, 0, stream>>>(
        N, gI, I, gO, offset_calculator,
        gI_stride * sizeof(scalar_t), I_stride * sizeof(scalar_t));
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });
}

Tensor& glu_backward_cuda_out(const Tensor& grad_output, const Tensor& input,
                              int64_t dim, Tensor& grad_input) {
  TORCH_CHECK(input.dim() > 0, "glu does not support 0-dimensional tensors");
  auto wrap_dim = maybe_wrap_dim(dim, input.dim());
  auto input_sizes = input.sizes();
  const int64_t nIn = input_sizes[wrap_dim];
  TORCH_CHECK(nIn % 2 == 0, "Halving dimension must be even, but dimension ",
              wrap_dim, " is size ", nIn);

  resize_output(grad_input, input_sizes);

  DimVector iter_shape(input_sizes);
  const auto dim_size = nIn / 2;
  iter_shape[wrap_dim] = dim_size;
  TORCH_CHECK(grad_output.sizes() == IntArrayRef{iter_shape});

  const auto iter = at::TensorIteratorConfig()
    .add_output(grad_input)
    .add_input(input)
    .add_input(grad_output)
    .resize_outputs(false)
    .declare_static_shape(iter_shape)
    .build();

  if (iter.numel() == 0) {
    return grad_input;
  }

  const auto I_stride = input.strides()[wrap_dim] * dim_size;
  const auto gI_stride = grad_input.strides()[wrap_dim] * dim_size;

  if (iter.can_use_32bit_indexing()) {
    launch_glu_backward_kernel(iter, gI_stride, I_stride);
  } else {
    for (auto sub_iter: iter.with_32bit_indexing()) {
      launch_glu_backward_kernel(sub_iter, gI_stride, I_stride);
    }
  }
  return grad_input;
}

Tensor glu_backward_cuda(const Tensor& grad_output, const Tensor& input, int64_t dim) {
  auto grad_input = at::empty({0}, input.options());
  return glu_backward_cuda_out(grad_output, input, dim, grad_input);
}

// -----------------------------------
// log_sigmoid forward
// -----------------------------------

std::tuple<Tensor&, Tensor&> log_sigmoid_forward_out_cuda(const Tensor& input, Tensor& result, Tensor& buffer) {
  // NOTE: buffer is only used by CPU dispatch, we just ignore it here
  auto iter = TensorIteratorConfig()
    .add_output(result)
    .add_input(input)
    .build();
  AT_DISPATCH_FLOATING_TYPES_AND(kHalf, iter.common_dtype(),
                                 "log_sigmoid_forward_cuda", [&] {
    using acc_t = acc_type<scalar_t, true>;
    gpu_kernel(iter,
        [] GPU_LAMBDA (scalar_t in_) -> scalar_t {
          const acc_t in = in_;
          const auto max = std::max(acc_t(0), -in);
          const auto z = std::exp(-max) + std::exp(-in - max);
          return -(max + std::log(z));
        });
  });
  return std::forward_as_tuple(result, buffer);
}

std::tuple<Tensor, Tensor> log_sigmoid_forward_cuda(const Tensor& input) {
  auto result = at::empty_like(input);
  auto buffer = at::empty({0}, input.options());
  log_sigmoid_forward_out_cuda(input, result, buffer);
  return std::forward_as_tuple(result, buffer);
}

// -----------------------------------
// log_sigmoid backward
// -----------------------------------

void log_sigmoid_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(kHalf, iter.common_dtype(),
                                 "log_sigmoid_backward_cuda", [&] {
    using acc_t = acc_type<scalar_t, true>;
    gpu_kernel(iter,
        [] GPU_LAMBDA (scalar_t in_, scalar_t grad_out_) -> scalar_t {
          const acc_t in = in_;
          const acc_t grad_out = grad_out_;
          const auto max = std::max(acc_t(0), -in);
          const auto z = std::exp(-max) + std::exp(-in - max);

          auto in_negative = in < acc_t(0);
          auto max_deriv = in_negative ? acc_t(1) : acc_t(0);
          auto sign = in_negative ? acc_t(1) : -acc_t(1);
          return grad_out * (max_deriv - sign * (acc_t(1) - acc_t(1) / z));
        });
  });
}

// -----------------------------------
// prelu forward
// -----------------------------------
template <typename scalar_t>
void prelu_cuda_kernel_share_weights(
  const Tensor& input,
  Tensor& result,
  const scalar_t* weight_data)
{
  auto iter = TensorIterator::unary_op(result, input);

  at::native::gpu_kernel(iter,
    [weight_data] GPU_LAMBDA (scalar_t input_val) {
        return (input_val > 0) ? input_val : *weight_data * input_val;
    });
}

template <typename scalar_t>
__global__ void prelu_cuda_kernel_multi_weights(
  scalar_t* result_data,
  const scalar_t* input_data,
  const scalar_t* weight_data,
  int64_t input_stride0,
  int64_t input_stride1,
  int64_t input_numel) {

  int64_t linearId = blockIdx.x * blockDim.x + threadIdx.x;
  if (linearId >= input_numel) return;

  // multiply values at each channel with weight[channel_index]
  int64_t channel = (linearId % input_stride0) / input_stride1;
  scalar_t input_data_val = input_data[linearId];
  result_data[linearId] = (input_data_val > 0) ? input_data_val : weight_data[channel] * input_data_val;
}

Tensor prelu_cuda(const Tensor& self, const Tensor& weight_) {
  TORCH_CHECK(self.is_cuda());
  TORCH_CHECK(weight_.is_cuda());

  auto input = self.contiguous();
  auto weight = weight_.contiguous();

  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(weight.is_contiguous());

  int64_t weight_num = weight.numel();
  Tensor result = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto strides = input.strides();

  // case1: shared weight for all channels
  if (weight_num == 1) {
    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_cuda", [&] {
      prelu_cuda_kernel_share_weights<scalar_t>(
        input,
        result,
        weight.data_ptr<scalar_t>());
    });
  }
  else { // case2: multiple weights, one for each channel
    int64_t input_ndim = input.dim();
    TORCH_CHECK(input_ndim > 0, "Not allow zero-dim input tensor.");

    int64_t channel_size = 1; // channel_size default to 1
    int64_t input_stride0 = 1, input_stride1 = 1;

    if (input_ndim > 1) {
      channel_size = input.size(1); // channel is the 2nd dim of input
      input_stride0 = strides[0];
      input_stride1 = strides[1];
    }
    TORCH_CHECK(channel_size == weight_num,
      "Mismatch of parameter numbers and input channel size. Found parameter numbers = ", weight_num,
      " and channel size = ", channel_size, ".");

    // config to run cuda kernel
    int64_t input_numel = input.numel();
    const dim3 block = dim3(std::min(static_cast<int64_t>(cuda::getApplyBlock().x), input_numel));
    dim3 grid;
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
    TORCH_CHECK(cuda::getApplyGrid(input_numel, grid, curDevice), "prelu: input too large or too many dimensions");

    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_cuda", [&] {
      prelu_cuda_kernel_multi_weights<scalar_t>
      <<<grid, block, 0, stream>>>(
        result.data_ptr<scalar_t>(),
        input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        input_stride0,
        input_stride1,
        input_numel);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
  }
  return result;
}

// -----------------------------------
// prelu backward
// -----------------------------------
template <typename scalar_t>
void prelu_cuda_backward_kernel_share_weights(
  const Tensor& input,
  const Tensor& grad_out,
  Tensor& input_grad,
  Tensor& weight_grad_collector,
  const scalar_t* weight_data) {
  at::TensorIterator iter = TensorIteratorConfig()
      .add_output(input_grad)
      .add_output(weight_grad_collector)
      .add_input(input)
      .add_input(grad_out)
      .build();

  // N.B. `std::tuple` does not support `::operator=` on device code.
  gpu_kernel_multiple_outputs(iter, [=] GPU_LAMBDA (scalar_t input, scalar_t grad_out) -> thrust::tuple<scalar_t, scalar_t> {
    scalar_t input_grad = input > 0 ? grad_out : (*weight_data) * grad_out;
    scalar_t weight_grad_collector = input > 0 ? scalar_t(0) : input * grad_out;
    return {input_grad, weight_grad_collector};
  });
}

template <typename scalar_t>
__global__ void prelu_cuda_backward_kernel_multi_weights(
  const scalar_t* input_data,
  const scalar_t* weight_data,
  const scalar_t* grad_out_data,
  scalar_t* input_grad_data,
  scalar_t* weight_grad_collector,
  int64_t input_stride0,
  int64_t input_stride1,
  int64_t input_numel) {

  int64_t linearId = blockIdx.x * blockDim.x + threadIdx.x;
  if (linearId >= input_numel) return;
  int64_t channel = (linearId % input_stride0) / input_stride1;
  scalar_t input_data_val = input_data[linearId];
  scalar_t grad_out_data_val = grad_out_data[linearId];
  input_grad_data[linearId] = (input_data_val > 0) ? grad_out_data_val : weight_data[channel] * grad_out_data_val;
  weight_grad_collector[linearId] = (input_data_val > 0) ? scalar_t(0) : input_data_val * grad_out_data_val;
}

std::tuple<Tensor, Tensor> prelu_backward_cuda(const Tensor& grad_out_, const Tensor& self, const Tensor& weight_) {
  TORCH_CHECK(grad_out_.is_cuda());
  TORCH_CHECK(self.is_cuda());
  TORCH_CHECK(weight_.is_cuda());

  auto input = self.contiguous();
  auto grad_out = grad_out_.contiguous();
  auto weight = weight_.contiguous();

  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(weight.is_contiguous());
  TORCH_CHECK(grad_out.is_contiguous());

  int64_t weight_num = weight.numel();
  auto strides = input.strides();
  auto dims = input.dim();
  Tensor input_grad = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor weight_grad = at::empty_like(weight, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor weight_grad_collector = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  // case1: shared parameter for all channels
  if (weight_num == 1) {
    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_backward_cuda", [&] {
      prelu_cuda_backward_kernel_share_weights<scalar_t>(
        input,
        grad_out,
        input_grad,
        weight_grad_collector,
        weight.data_ptr<scalar_t>());
    });
    weight_grad.fill_(weight_grad_collector.sum());
  }
  else { // case2: multiple parameters, one for each channel
    int64_t input_ndim = input.dim();
    TORCH_CHECK(input_ndim > 0, "Not allow zero-dim input tensor.");

    int64_t channel_size = 1; // channel_size default to 1
    int64_t input_stride0 = 1, input_stride1 = 1;

    if (input_ndim > 1) {
      channel_size = input.size(1); // channel is the 2nd dim of input
      input_stride0 = strides[0];
      input_stride1 = strides[1];
    }
    TORCH_CHECK(channel_size == weight_num,
      "Mismatch of parameter numbers and input channel size. Found parameter numbers = ", weight_num,
      " and channel size = ", channel_size, ".");

    // config to run cuda kernel
    int64_t input_numel = input.numel();
    const dim3 block = dim3(std::min(static_cast<int64_t>(cuda::getApplyBlock().x), input_numel));
    dim3 grid;
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
    TORCH_CHECK(cuda::getApplyGrid(input_numel, grid, curDevice), "prelu_backward_cuda: input too large or too many dimensions");

    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_backward_cuda", [&] {
      prelu_cuda_backward_kernel_multi_weights<scalar_t>
      <<<grid, block, 0, stream>>>(
        input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        grad_out.data_ptr<scalar_t>(),
        input_grad.data_ptr<scalar_t>(),
        weight_grad_collector.data_ptr<scalar_t>(),
        input_stride0,
        input_stride1,
        input_numel);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
    // update weight_grad
    std::vector<int64_t> reduce_dims;
    reduce_dims.push_back(0);
    if (dims > 2) {
      for(int64_t i = 2; i < dims; i++) reduce_dims.push_back(i);
    }
    weight_grad = weight_grad_collector.sum(reduce_dims);
  }
  return std::tuple<Tensor, Tensor>{input_grad, weight_grad};
}

// -----------------------------------
// rrelu
// -----------------------------------
template <typename scalar_t, int unroll_factor, typename F>
#if __CUDA_ARCH__ >= 350 || defined __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_2(256, 4)
#endif
__global__ void rrelu_with_noise_cuda_kernel(
    int numel,
    PhiloxCudaState philox_args,
    scalar_t* output,
    scalar_t* input,
    scalar_t* noise,
    double lower,
    double upper,
    const F& random_func) {
  auto seeds = at::cuda::philox::unpack(philox_args);
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(std::get<0>(seeds),
              idx,
              std::get<1>(seeds),
              &state);

  int grid_stride = blockDim.x * gridDim.x * unroll_factor;
  int rounded_size = ((numel - 1) / grid_stride + 1) * grid_stride;
  double range = upper - lower;

  for (int linear_index = idx; linear_index < rounded_size; linear_index += grid_stride) {
    auto rand = random_func(&state);

    // ensure that (&rand.x)[ii] is safe
    static_assert(sizeof(rand)/sizeof(rand.x) == unroll_factor, "");

    #pragma unroll
    for (int ii = 0; ii < unroll_factor; ii++) {
      int li = linear_index + blockDim.x * gridDim.x * ii;
      if (li >= numel) {
        continue;
      }
      scalar_t r = static_cast<scalar_t>((&rand.x)[ii]);
      r = r * range + lower;
      if (input[li] <= 0) {
        output[li] = input[li] * r;
        noise[li] = r;
      } else {
        output[li] = input[li];
        noise[li] = static_cast<scalar_t>(0);
      }
    }
    __syncthreads();
  }
}

template <typename scalar_t>
inline void _rrelu_with_noise_cuda_train(
    Tensor& output,
    const Tensor& input_,
    const Tensor& noise_,
    const Scalar& lower_,
    const Scalar& upper_,
    c10::optional<Generator> generator) {
  auto input = input_.contiguous();
  auto noise = noise_.contiguous();
  Tensor tmp_output = output.contiguous();

  int64_t numel = input.numel();
  auto execution_policy = calc_execution_policy(numel);

  auto counter_offset = std::get<0>(execution_policy);
  auto grid = std::get<1>(execution_policy);
  auto block = std::get<2>(execution_policy);

  auto gen = get_generator_or_default<CUDAGeneratorImpl>(
      generator, cuda::detail::getDefaultCUDAGenerator());
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(counter_offset);
  }

  scalar_t* input_data = input.data_ptr<scalar_t>();
  scalar_t* noise_data = noise.data_ptr<scalar_t>();
  scalar_t* output_data = tmp_output.data_ptr<scalar_t>();

  double lower = lower_.to<double>();
  double upper = upper_.to<double>();

  auto stream = at::cuda::getCurrentCUDAStream();

  if (std::is_same<scalar_t, double>::value) {
    rrelu_with_noise_cuda_kernel<scalar_t, 2><<<grid, block, 0, stream>>>(
        numel,
        rng_engine_inputs,
        output_data,
        input_data,
        noise_data,
        lower,
        upper,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) {
          return hiprand_uniform2_double(state);
        });
        C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    // half and float
    rrelu_with_noise_cuda_kernel<scalar_t, 4><<<grid, block, 0, stream>>>(
        numel,
        rng_engine_inputs,
        output_data,
        input_data,
        noise_data,
        lower, upper,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) {
          return hiprand_uniform4(state);
        });
        C10_CUDA_KERNEL_LAUNCH_CHECK();
  }

  if (!output.is_contiguous()) {
    output.copy_(tmp_output);
  }
}

Tensor& rrelu_with_noise_out_cuda(const Tensor& self,
    const Tensor& noise,
    const Scalar& lower,
    const Scalar& upper,
    bool training,
    c10::optional<Generator> generator,
    Tensor& output) {
  TensorArg self_arg{self, "self", 1}, noise_arg{noise, "noise", 2},
      output_arg{output, "output", 3};
  checkAllSameGPU("rrelu_with_noise_out_cuda", {self_arg, noise_arg, output_arg});

  if (training) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        self.scalar_type(), "rrelu_with_noise_out_cuda", [&] {
          _rrelu_with_noise_cuda_train<scalar_t>(
              output, self, noise, lower, upper, generator);
        });
  }
  else {
    auto lower_tensor = lower.to<double>();
    auto upper_tensor = upper.to<double>();
    Scalar negative_slope = (lower_tensor + upper_tensor) / 2;
    at::leaky_relu_out(output, self, negative_slope);
  }
  return output;
}

Tensor rrelu_with_noise_cuda(
    const Tensor& self,
    const Tensor& noise,
    const Scalar& lower,
    const Scalar& upper,
    bool training,
    c10::optional<Generator> generator) {
  Tensor output = at::empty_like(self, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  return at::native::rrelu_with_noise_out_cuda(self, noise, lower, upper, training, generator, output);
}

Tensor& rrelu_with_noise_cuda_(
    Tensor& self,
    const Tensor& noise,
    const Scalar& lower,
    const Scalar& upper,
    bool training,
    c10::optional<Generator> generator) {
  return at::native::rrelu_with_noise_out_cuda(
      self, noise, lower, upper, training, generator, self);
}

// -----------------------------------
// hardshrink
// -----------------------------------
void hardshrink_kernel(TensorIteratorBase& iter, const Scalar& value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardshrink_cuda", [&]() {
    auto lambd = value.to<scalar_t>();
    gpu_kernel(iter, [lambd]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return (a >= -lambd && a <= lambd) ? scalar_t(0) : a;
    });
  });
}

void softshrink_kernel(TensorIteratorBase& iter, const Scalar& value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "softshrink_cuda", [&]() {
    auto lambd = value.to<scalar_t>();
    gpu_kernel(iter, [lambd]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a > lambd ? a - lambd : (a < -lambd ? a + lambd : scalar_t(0));
    });
  });
}

void shrink_backward_kernel(TensorIteratorBase& iter, const Scalar& value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "shrink_backward_cuda", [&]() {
    auto lambd = value.to<scalar_t>();
    gpu_kernel(iter, [lambd]GPU_LAMBDA(scalar_t grad_val, scalar_t self_val) -> scalar_t {
      return (self_val >= -lambd && self_val <= lambd) ? scalar_t(0) : grad_val;
    });
  });
}

void hardtanh_backward_kernel(TensorIterator& iter, const Scalar& min, const Scalar& max) {
  AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, iter.dtype(), "hardtanh_backward_cuda", [&]() {
    auto min_val = min.to<scalar_t>();
    auto max_val = max.to<scalar_t>();
    gpu_kernel(iter, [min_val, max_val]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return (b <= min_val) || (b >= max_val) ? scalar_t(0) : a;
    });
  });
}

void softplus_kernel(TensorIteratorBase& iter, const Scalar& beta_, const Scalar& threshold_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "softplus_cuda", [&]() {
    auto beta = beta_.to<scalar_t>();
    auto threshold = threshold_.to<scalar_t>();
    gpu_kernel(iter, [beta, threshold]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return (a * beta) > threshold ? a : static_cast<scalar_t>(::log1p(std::exp(a * beta))) / beta;
    });
  });
}

void softplus_backward_kernel(TensorIteratorBase& iter, const Scalar& beta_, const Scalar& threshold_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "softplus_backward_cuda", [&]() {
    auto beta = beta_.to<scalar_t>();
    auto threshold = threshold_.to<scalar_t>();
    gpu_kernel(iter, [beta, threshold]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      scalar_t z = std::exp(b * beta);
      return (b * beta) > threshold ? a : a * z / (z + scalar_t(1.));
    });
  });
}

template <typename scalar_t>
void threshold_kernel_impl(TensorIteratorBase& iter, scalar_t threshold, scalar_t value) {
  gpu_kernel_with_scalars(iter, [=]GPU_LAMBDA(scalar_t x, scalar_t other) -> scalar_t {
    return x <= threshold ? value : other;
  });
}

static void threshold_kernel_cuda(TensorIteratorBase& iter, const Scalar& threshold, const Scalar& value) {
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "threshold_cuda", [&] {
    threshold_kernel_impl<scalar_t>(iter, threshold.to<scalar_t>(), value.to<scalar_t>());
  });
}

void elu_kernel(TensorIteratorBase& iter, const Scalar& alpha, const Scalar& scale, const Scalar& input_scale) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "elu_cuda", [&]() {
    auto negcoef = alpha.to<scalar_t>() * scale.to<scalar_t>();
    auto poscoef = scale.to<scalar_t>();
    auto negiptcoef = input_scale.to<scalar_t>();
    gpu_kernel(iter, [negcoef, poscoef, negiptcoef]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a > scalar_t(0) ? a * poscoef : (static_cast<scalar_t>(std::exp(a * negiptcoef)) - scalar_t(1.)) * negcoef;
    });
  });
}

void elu_backward_kernel(TensorIteratorBase& iter, const Scalar& alpha, const Scalar& scale, const Scalar& input_scale, bool is_result) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "elu_backward_cuda", [&]() {
    auto negcoef = alpha.to<scalar_t>() * scale.to<scalar_t>();
    auto poscoef = scale.to<scalar_t>();
    auto negiptcoef = input_scale.to<scalar_t>();
    gpu_kernel(iter, [negcoef, poscoef, negiptcoef, is_result]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      if (is_result) {
        return b <= scalar_t(0) ? a * negiptcoef * (b + negcoef) : a * poscoef;
      } else {
        return b <= scalar_t(0) ? a * negiptcoef * negcoef * (static_cast<scalar_t>(std::exp(b * negiptcoef))) : a * poscoef;
      }
    });
  });
}

namespace {

void GeluCUDAKernelImpl(TensorIteratorBase& it) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, it.dtype(), "GeluCUDAKernelImpl", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    gpu_kernel(it, [] GPU_LAMBDA(scalar_t x) -> scalar_t {
      return static_cast<T_ACC>(x) *
          c10::cuda::compat::normcdf(static_cast<T_ACC>(x));
    });
  });
}

void GeluBackwardCUDAKernelImpl(TensorIteratorBase& it) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
      it.dtype(), "GeluBackwardCUDAKernelImpl", [&]() {
        using T_ACC = acc_type<scalar_t, true>;
        gpu_kernel(it, [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
          constexpr T_ACC kBeta = M_2_SQRTPI * M_SQRT1_2 * T_ACC(0.5);
          const T_ACC cdf = c10::cuda::compat::normcdf(static_cast<T_ACC>(x));
          const T_ACC pdf =
              c10::cuda::compat::exp(
                  T_ACC(-0.5) * static_cast<T_ACC>(x) * static_cast<T_ACC>(x)) *
              kBeta;
          return static_cast<T_ACC>(dy) * (cdf + static_cast<T_ACC>(x) * pdf);
        });
      });
}

void leaky_relu_kernel(TensorIteratorBase& iter, const Scalar& negval_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "leaky_relu_cuda", [&]() {
    auto negval = negval_.to<scalar_t>();
    gpu_kernel(iter, [negval]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a > scalar_t(0) ? a : a * negval;
    });
  });
}

void leaky_relu_backward_kernel(TensorIteratorBase& iter, const Scalar& negval_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "leaky_relu_backward_cuda", [&]() {
    auto negval = negval_.to<scalar_t>();
    gpu_kernel(iter, [negval]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a > scalar_t(0) ? b : b * negval;
    });
  });
}

void hardswish_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardswish_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC one_sixth(1.0f / 6.0f);
    const T_ACC three(3.0f);
    const T_ACC six(6.0f);
    gpu_kernel(iter, [zero, one_sixth, three, six]GPU_LAMBDA(scalar_t self_val) -> scalar_t {
      T_ACC x = static_cast<T_ACC>(self_val);
      return x * std::min(std::max(x + three, zero), six) * one_sixth;
    });
  });
}

void hardswish_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardswish_backward_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC three(3.0f);
    const T_ACC neg_three(-3.0f);
    const T_ACC one_half(0.5f);
    gpu_kernel(
      iter,
      [zero, three, neg_three, one_half]GPU_LAMBDA(scalar_t grad_val_, scalar_t self_val_) -> scalar_t {
        T_ACC grad_val = static_cast<T_ACC>(grad_val_);
        T_ACC self_val = static_cast<T_ACC>(self_val_);
        if (self_val < neg_three) {
          return zero;
        } else if (self_val <= three) {
          return grad_val * ((self_val / three) + one_half);
        } else {
          return grad_val;
        }
    });
  });
}

void hardsigmoid_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardsigmoid_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC one_sixth(1.0f / 6.0f);
    const T_ACC three(3.0f);
    const T_ACC six(6.0f);
    gpu_kernel(iter, [zero, one_sixth, three, six]GPU_LAMBDA(scalar_t self_val) -> scalar_t {
      T_ACC x = static_cast<T_ACC>(self_val);
      return std::min(std::max(x + three, zero), six) * one_sixth;
    });
  });
}

void hardsigmoid_backward_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardsigmoid_backward_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC three(3.0f);
    const T_ACC neg_three(-3.0f);
    const T_ACC one_sixth(1.0f / 6.0f);
    gpu_kernel(
      iter,
      [zero, three, neg_three, one_sixth]GPU_LAMBDA(scalar_t grad_val_, scalar_t self_val_) -> scalar_t {
        T_ACC grad_val = static_cast<T_ACC>(grad_val_);
        T_ACC self_val = static_cast<T_ACC>(self_val_);
        return (self_val > neg_three && self_val < three)
          ? grad_val * one_sixth
          : zero;
    });
  });
}

void silu_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "silu_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t x) -> scalar_t {
              using T_ACC = acc_type<scalar_t, true>;
              const T_ACC x_acc = static_cast<T_ACC>(x);
              return x_acc / (T_ACC(1) + c10::cuda::compat::exp(-x_acc));
            });
      });
}

void silu_backward_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "silu_backward_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
              using T_ACC = acc_type<scalar_t, true>;
              const T_ACC dy_acc = static_cast<T_ACC>(dy);
              const T_ACC x_acc = static_cast<T_ACC>(x);
              const T_ACC s_acc =
                  T_ACC(1) / (T_ACC(1) + c10::cuda::compat::exp(-x_acc));
              return dy_acc * s_acc * (T_ACC(1) + x_acc * (T_ACC(1) - s_acc));
            });
      });
}

void mish_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "mish_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t x) -> scalar_t {
          using T_ACC = acc_type<scalar_t, true>;
          const T_ACC x_acc = static_cast<T_ACC>(x);
          return x_acc * c10::cuda::compat::tanh(c10::cuda::compat::log1p(c10::cuda::compat::exp(x_acc)));
      });
      });
}

void mish_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "mish_backward_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
          using T_ACC = acc_type<scalar_t, true>;
          const T_ACC dy_acc = static_cast<T_ACC>(dy);
          const T_ACC x_acc = static_cast<T_ACC>(x);
          const T_ACC s_acc =
              T_ACC(1) / (T_ACC(1) + c10::cuda::compat::exp(-x_acc));
          const T_ACC t_acc =
              c10::cuda::compat::tanh(c10::cuda::compat::log1p(c10::cuda::compat::exp(x_acc)));
          return dy_acc * (t_acc + x_acc * s_acc * (T_ACC(1) - t_acc * t_acc));
      });
      });
}

} // namespace

TORCH_IMPL_FUNC(gelu_out_cuda) (
  const Tensor& self, const Tensor& result
) {
  GeluCUDAKernelImpl(*this);
}

TORCH_IMPL_FUNC(gelu_backward_out_cuda) (
  const Tensor& grad, const Tensor& self, const Tensor& grad_input
) {
  GeluBackwardCUDAKernelImpl(*this);
}

REGISTER_DISPATCH(hardtanh_backward_stub, &hardtanh_backward_kernel);
REGISTER_DISPATCH(hardshrink_stub, &hardshrink_kernel);
REGISTER_DISPATCH(log_sigmoid_backward_stub, &log_sigmoid_backward_kernel);
REGISTER_DISPATCH(softshrink_stub, &softshrink_kernel);
REGISTER_DISPATCH(shrink_backward_stub, &shrink_backward_kernel);
REGISTER_DISPATCH(elu_stub, &elu_kernel);
REGISTER_DISPATCH(elu_backward_stub, &elu_backward_kernel);
REGISTER_DISPATCH(glu_stub, &glu_kernel);
REGISTER_DISPATCH(leaky_relu_stub, &leaky_relu_kernel);
REGISTER_DISPATCH(leaky_relu_backward_stub, &leaky_relu_backward_kernel);
REGISTER_DISPATCH(hardswish_stub, &hardswish_kernel);
REGISTER_DISPATCH(hardswish_backward_stub, &hardswish_backward_kernel);
REGISTER_DISPATCH(hardsigmoid_stub, &hardsigmoid_kernel);
REGISTER_DISPATCH(hardsigmoid_backward_stub, &hardsigmoid_backward_kernel);
REGISTER_DISPATCH(softplus_stub, &softplus_kernel);
REGISTER_DISPATCH(softplus_backward_stub, &softplus_backward_kernel);
REGISTER_DISPATCH(silu_stub, &silu_kernel);
REGISTER_DISPATCH(silu_backward_stub, &silu_backward_kernel);
REGISTER_DISPATCH(mish_stub, &mish_kernel);
REGISTER_DISPATCH(mish_backward_stub, &mish_backward_kernel);
REGISTER_DISPATCH(threshold_stub, &threshold_kernel_cuda);

} // namespace native
} // namespace at
