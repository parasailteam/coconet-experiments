#include <ATen/ATen.h>
#include <ATen/native/cuda/SortingCommon.cuh>

#include <THC/THCThrustAllocator.cuh>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/device_ptr.h>

namespace at { namespace native {

void index_put_with_sort_kernel_thrust_helper(Tensor &linearIndex, Tensor &orig_indices, Tensor &sorted_indices, int64_t num_indices) {
  sorted_indices.copy_(linearIndex);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  using device_ptr = thrust::device_ptr<int64_t>;

  // Fill sortedOrigIndices with sequential indices
  const auto count_iter = thrust::counting_iterator<int64_t>(0);
  auto orig_data = device_ptr(orig_indices.data_ptr<int64_t>());
  thrust::copy(policy, count_iter, count_iter + num_indices, orig_data);

  // Sort the inputs into sorted with the corresponding indices; we
  // don't need a stable or multidimensional sort, so just use Thrust
  // directly
  // Sort; a stable sort is not required
  // NB - not passing comparator causes thrust to use radix sort, and it hurts perf A LOT, at least for medium (few K) sized indices
  auto sorted_data = device_ptr(sorted_indices.data_ptr<int64_t>());
  thrust::sort_by_key(policy, sorted_data, sorted_data + num_indices, orig_data, LTOp<int64_t>());
}

template<typename index_t>
void embedding_dense_backward_cuda_scan(Tensor &sorted_indices, Tensor &count) {
  using device_ptr = thrust::device_ptr<index_t>;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  auto num_indices = count.numel();

  // Compute an increasing sequence per unique item in sortedIndices:
  // sorted: 2 5 5 5 7 7 8 9 9
  //  count: 1 1 2 3 1 2 1 1 2
  auto sorted_data = device_ptr(sorted_indices.data_ptr<index_t>());
  auto count_data = device_ptr(count.data_ptr<index_t>());
  thrust::inclusive_scan_by_key(
    policy,
    sorted_data,
    sorted_data + num_indices,
    thrust::make_constant_iterator(1),
    count_data
  );

  // Take the maximum of each count per unique key in reverse:
  // sorted: 2 5 5 5 7 7 8 9 9
  //  count: 1 3 3 3 2 2 1 2 2
  thrust::inclusive_scan_by_key(
    policy,
    thrust::make_reverse_iterator(sorted_data + num_indices),
    thrust::make_reverse_iterator(sorted_data),
    thrust::make_reverse_iterator(count_data + num_indices),
    thrust::make_reverse_iterator(count_data + num_indices),
    thrust::equal_to<index_t>(),
    thrust::maximum<index_t>()
  );
}

template
void embedding_dense_backward_cuda_scan<int>(Tensor &sorted_indices, Tensor &count);
template
void embedding_dense_backward_cuda_scan<int64_t>(Tensor &sorted_indices, Tensor &count);

}}
